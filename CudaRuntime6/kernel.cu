#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <chrono>

using namespace std;

hipError_t mergeSortCuda(int* table_in, int* table_out, unsigned long int size);

void generateNumbers(int* table1, int* table2, unsigned long int n);
void print_table(int* table, unsigned long int table_size);
int mergeCPU(int* table_in1, int* table_in2, int* table_in, unsigned long int len);
int mergeSortCPU(int* table_in, unsigned long int len);
void myFunction(int* myTable, int size);


__global__ void MaclaurinSeriesKernel(int* array, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        float x = static_cast<float>(array[i]) / 1000.0;  // Adjust x for better convergence
        float result = 1.0;
        float term = 1.0;
        for (int j = 1; j <= 100; j++) {
            term *= x / j;
            result += term;
        }
        array[i] = static_cast<int>(1000.0 * result);
    }
}


void MaclaurinSeries(int* array, int size) {
    for (int i = 0; i < size; i++) {
        float x = static_cast<float>(array[i]) / 1000.0;  // Adjust x for better convergence
        float result = 1.0;
        float term = 1.0;
        for (int j = 1; j <= 100; j++) {
            term *= x / j;
            result += term;
        }
        array[i] = static_cast<int>(1000.0 * result);
    }
}

__global__ void ReverseSortKernel(int* array, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size / 2) {
        int temp = array[i];
        array[i] = array[size - 1 - i];
        array[size - 1 - i] = temp;
    }
}

void ReverseSort(int* array, int size) {
    for (int i = 0; i < size / 2; i++) {
        int temp = array[i];
        array[i] = array[size - 1 - i];
        array[size - 1 - i] = temp;
    }
}



void myFunction(int* myTable, int size) {
    printf("\n");
    printf("len of table = %d\n", size);
    int* buffor = new int[size];

    for (int i = 2048; i <= size; i *= 2) {
        int iterations = size / i;
        for (int j = 0; j < iterations; j++) {
            int start = 0 + j * i;
            int middle = i / 2 + start;
            int startStop = middle;
            int middleStop = i + start;
            for (int k = 0 + i * j; k < i * j + i; k++) {
                if (start < startStop) {
                    if (middle < middleStop) {

                        if (myTable[start] < myTable[middle])
                        {
                            buffor[k] = myTable[start++];
                        }
                        else
                        {
                            buffor[k] = myTable[middle++];
                        }
                    }
                    else
                    {
                        buffor[k] = myTable[start++];

                    }
                }
                else
                {
                    buffor[k] = myTable[middle++];
                }

            }

        }
        for (int l = 0; l < size; l++) { myTable[l] = buffor[l]; }
    }

    delete[] buffor;
}

__global__ void mergeKernelShared2(int* table_in, int* table_out) {
    extern __shared__ int sdata[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int si = threadIdx.x;

    sdata[si] = table_in[i];
    __syncthreads();


    for (unsigned int lvl = 2; lvl <= blockDim.x; lvl *= 2) {
        if (si < blockDim.x / lvl) {
            int idx = threadIdx.x * lvl;
            int start = threadIdx.x * lvl;
            int middle = start + lvl / 2;
            int startStop = middle;
            int middleStop = startStop + lvl / 2;
#pragma unroll
            for (unsigned z = 0; z < lvl; z++) {

                if (start < startStop && middle < middleStop) {
                    if (sdata[start] < sdata[middle]) {
                        sdata[blockDim.x + idx + z] = sdata[start++];
                    }
                    else {
                        sdata[blockDim.x + idx + z] = sdata[middle++];
                    }
                }
                else {
                    if (start < startStop) {
                        sdata[blockDim.x + idx + z] = sdata[start++];
                    }
                    else
                    {
                        sdata[blockDim.x + idx + z] = sdata[middle++];
                    }
                }
            }
            for (unsigned b = threadIdx.x * lvl; b < threadIdx.x * lvl + lvl; b++) {
                sdata[b] = sdata[blockDim.x + b];

            }
        }
        __syncthreads();

    }

    table_out[i] = sdata[si];
}


__global__ void mergeKernel(int* table_in, int* table_out, unsigned long int lvl) {
    unsigned long int i = (blockIdx.x * blockDim.x + threadIdx.x) * lvl;

    unsigned long int start = i;
    unsigned long int middle = i + lvl / 2;
    unsigned long int start1 = i;
    unsigned long int middle1 = i + lvl / 2;
    for (unsigned long int j = i; j < i + lvl; j++) {
        if (start < middle1 && middle < start1 + lvl) {
            if (table_in[start] < table_in[middle]) {
                table_out[j] = table_in[start];
                start++;
            }
            else {
                table_out[j] = table_in[middle];
                middle++;
            }
        }
        else {
            if (start < middle1) {
                table_out[j] = table_in[start];
                start++;
            }
            else {
                table_out[j] = table_in[middle];
                middle++;
            }
        }
    }
}

void generateNumbers(int* table1, int* table2, unsigned long int n)
{
    int random_number = 0;
    for (int i = 0; i < n; i++) {
        random_number = rand() % 1000 + 1;
        table1[i] = random_number;
        table2[i] = random_number;
    }
}

__global__ void dummyKernel() {
    // This kernel does nothing but ensures the GPU is initialized.
}

int mergeCPU(int* table_in1, int* table_in2, int* table_in, unsigned long int len) {
    unsigned long int start = 0;
    unsigned long int start1 = 0;
    unsigned long int middle = 0;
    unsigned long int middle1 = len;
    unsigned long int size = middle1 * 2;
    for (unsigned long int j = 0; j < size; j++) {
        if (start < middle1 && middle < middle1) {
            if (table_in1[start] < table_in2[middle]) {
                table_in[j] = table_in1[start];
                start++;
            }
            else {
                table_in[j] = table_in2[middle];
                middle++;
            }
        }
        else {
            if (start < middle1) {
                table_in[j] = table_in1[start];
                start++;
            }
            else {
                table_in[j] = table_in2[middle];
                middle++;
            }

        }

    }

    return *table_in;
}


int mergeSortCPU(int* table_in, unsigned long int len) {

    unsigned long int table_length = len;
    unsigned long int half_length = table_length / 2;
    if (table_length <= 1) {
        return *table_in;
    }

    int* first_half = new int[half_length];
    int* second_half = new int[half_length];
    for (int i = 0; i < half_length; i++) {
        first_half[i] = table_in[i];
        second_half[i] = table_in[i + half_length];
    }

    *first_half = mergeSortCPU(first_half, half_length);
    *second_half = mergeSortCPU(second_half, half_length);
    //print_table(first_half, half_length);
    //print_table(second_half, half_length);
    *table_in = mergeCPU(first_half, second_half, table_in, half_length);
    //print_table(table_in, table_length);
    delete[] first_half;
    delete[] second_half;

    return *table_in;
}

void print_table(int* table, unsigned long int table_size) {
    printf("table = { ");
    for (unsigned long int i = 0; i < table_size - 1; i++)
        printf("%d, ", table[i]);
    printf("%d}\n", table[table_size - 1]);
}

int main() {
    dummyKernel << <1, 1 >> > ();
    hipDeviceSynchronize();
    const int MAX_THREADS = 1024;

    unsigned long int dynamic_size = 33554432;

    int* dynamic_table = new int[dynamic_size];
    int* dynamic_table_CPU = new int[dynamic_size];
    int* dynamic_table_out = new int[dynamic_size];

    generateNumbers(dynamic_table, dynamic_table_CPU, dynamic_size);

    auto start_CPU = chrono::high_resolution_clock::now();
    mergeSortCPU(dynamic_table_CPU, dynamic_size);
    auto end_CPU = chrono::high_resolution_clock::now();
    chrono::duration<float> duration_CPU = end_CPU - start_CPU;

    std::cout << "CPU Sort Time: " << duration_CPU.count() << " seconds\n";

    auto start_GPU = chrono::high_resolution_clock::now();
    hipError_t cudaStatus1 = mergeSortCuda(dynamic_table, dynamic_table_out, dynamic_size);
    auto end_GPU = chrono::high_resolution_clock::now();
    chrono::duration<float> duration_GPU = end_GPU - start_GPU;

    std::cout << "GPU Sort Time: " << duration_GPU.count() << " seconds\n";

    

   

    



    // Display a few elements from the arrays
    int step = 1;
    printf("CPU: ");
    for (int i = 0; i < 16; i += 1) {
        printf("%d ", dynamic_table_CPU[i * step]);
    }
    printf("%d %d", dynamic_table_CPU[dynamic_size / 2], dynamic_table_CPU[dynamic_size - 1]);
    printf("\nGPU: ");
    for (int i = 0; i < 16; i += 1) {
        printf("%d ", dynamic_table_out[i * step]);
    }
    printf("%d %d", dynamic_table_out[dynamic_size / 2], dynamic_table_out[dynamic_size - 1]);



    auto start_CPU_math = chrono::high_resolution_clock::now();
    MaclaurinSeries(dynamic_table_CPU, dynamic_size);
    auto end_CPU_math = chrono::high_resolution_clock::now();
    chrono::duration<float> duration_CPU_math = end_CPU_math - start_CPU_math;

    std::cout << "\nCPU MaclaurinSeries Operation Time: " << duration_CPU_math.count() << " seconds\n";

    
    auto start_GPU_math = chrono::high_resolution_clock::now();
    MaclaurinSeriesKernel << <(dynamic_size + 1023) / 1024, 1024 >> > (dynamic_table_out, dynamic_size);
    hipDeviceSynchronize();
    auto end_GPU_math = chrono::high_resolution_clock::now();
    chrono::duration<float> duration_GPU_math = end_GPU_math - start_GPU_math;

    std::cout << "GPU MaclaurinSeries Operation Time: " << duration_GPU_math.count() << " seconds\n";

   


  
    auto start_CPU_reverse = chrono::high_resolution_clock::now();
    ReverseSort(dynamic_table_CPU, dynamic_size);
    auto end_CPU_reverse = chrono::high_resolution_clock::now();
    chrono::duration<float> duration_CPU_reverse = end_CPU_reverse - start_CPU_reverse;

    std::cout << "CPU ReverseSort Time: " << duration_CPU_reverse.count() << " seconds\n";




    auto start_GPU_reverse = chrono::high_resolution_clock::now();
    ReverseSortKernel << <(dynamic_size + 1023) / 1024, 1024 >> > (dynamic_table_out, dynamic_size);
    hipDeviceSynchronize();
    auto end_GPU_reverse = chrono::high_resolution_clock::now();
    chrono::duration<float> duration_GPU_reverse = end_GPU_reverse - start_GPU_reverse;

    std::cout << "GPU ReverseSort Time: " << duration_GPU_reverse.count() << " seconds\n";

    



    // Clean up
    delete[] dynamic_table;
    delete[] dynamic_table_CPU;
    delete[] dynamic_table_out;

    cudaStatus1 = hipDeviceReset();
    if (cudaStatus1 != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t mergeSortCuda(int* table_in, int* table_out, unsigned long int size) {

    int* device_table_in = 0;
    int* device_table_in2 = 0;
    int* device_table_out = 0;
    int* device_table_out2 = 0;
    unsigned long int number_of_blocks = 1;
    unsigned long int number_of_threads = size;

    unsigned long int halfSize = size / 2;
    size_t halfSizeInBytes = halfSize * sizeof(int);
    int* secondHalfDevicePtr = device_table_in + halfSize;
    int* secondHalfHostPtr = device_table_out + halfSize;


    hipError_t cudaStatus;
    hipStream_t stream1, stream2;

    hipError_t r1 = hipStreamCreate(&stream1);
    hipError_t r2 = hipStreamCreate(&stream2);
    auto start_k = chrono::high_resolution_clock::now();

    //now choose device
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&device_table_out, size / 2 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 11");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&device_table_in, size / 2 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 21");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&device_table_out2, size / 2 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 12");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&device_table_in2, size / 2 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 22");
        goto Error;
    }
    auto end_k = chrono::high_resolution_clock::now();
    chrono::duration<float> duration_k = end_k - start_k;
    std::cout << " | data alocation Time = " << duration_k.count() << endl;


    cudaStatus = hipMemcpyAsync(device_table_in, table_in, (size / 2) * sizeof(int), hipMemcpyHostToDevice, stream1);
    mergeKernelShared2 << < (size / 1024) / 2, 1024, 1024 * 2 * 4, stream1 >> > (device_table_in, device_table_out);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "1 kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    cudaStatus = hipMemcpyAsync(device_table_in2, table_in + halfSize, (size / 2) * sizeof(int), hipMemcpyHostToDevice, stream2);
    mergeKernelShared2 << < (size / 1024) / 2, 1024, 1024 * 2 * 4, stream2 >> > (device_table_in2, device_table_out2);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "2 kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        goto Error;
    }

    /*
    cudaStatus = hipMemcpy(table_out, device_table_out, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    */

    cudaStatus = hipMemcpyAsync(table_out, device_table_out, (size / 2) * sizeof(int), hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpyAsync(table_out + halfSize, device_table_out2, (size / 2) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    table_in = table_out;
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

Error:
    hipFree(device_table_in);
    hipFree(device_table_out);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    return cudaStatus;
}
